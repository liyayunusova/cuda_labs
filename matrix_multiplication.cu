#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float* d_A, float* d_B, float* d_C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += d_A[row * N + k] * d_B[k * N + col];
        }
        d_C[row * N + col] = sum;
    }
}

// Function to fill matrix with random numbers
void fillMatrix(float* matrix, int N) {
    for (int i = 0; i < N * N; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

// Function to print the matrix
void printMatrix(const float* matrix, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << matrix[i * N + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    srand(time(0));

    int N;
    std::cout << "Enter the size of the matrix: ";
    std::cin >> N;

    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Fill matrices with random numbers
    fillMatrix(h_A, N);
    fillMatrix(h_B, N);

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Start timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the matrix multiplication kernel
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Stop timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the time taken
    std::cout << "Time taken for matrix multiplication: " << milliseconds << " ms" << std::endl;

    // Print the resulting matrix (optional)
    // printMatrix(h_C, N);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
